#include "job.cuh"

#include "common.cuh"

int calc_jobs(int real_job_num)
{
  return (real_job_num / THREADS_PER_BLOCK) * THREADS_PER_BLOCK + real_job_num % THREADS_PER_BLOCK;
}

job_t allocate_host_job(job_t job)
{
  job_t host_job = job;
  int jobs_num = calc_jobs(host_job.image_width * host_job.image_height);
  safeMalloc((void **) &host_job.gather_arr, sizeof(int) * jobs_num);
  safeMalloc((void **)&host_job.target_idx, sizeof(int) * jobs_num);
  safeMalloc((void **)&host_job.image_dest, sizeof(float) * jobs_num);
  safeMalloc((void **)&host_job.ray_pos, sizeof(float) * jobs_num);
  safeMalloc((void **)&host_job.ray_dir, sizeof(float) * jobs_num);

  return host_job;
}

void free_host_job(job_t *host_job)
{
  safeFree(host_job->gather_arr);
  host_job->gather_arr = NULL;
  safeFree(host_job->target_idx);
  host_job->target_idx = NULL;
  safeFree(host_job->image_dest);
  host_job->image_dest = NULL;
  safeFree(host_job->ray_pos);
  host_job->ray_pos = NULL;
  safeFree(host_job->ray_dir);
  host_job->ray_dir = NULL;
}

job_t allocate_device_job(job_t job)
{
  job_t dev_job = job;
  int jobs_num = calc_jobs(dev_job.image_width * dev_job.image_height);
  cudaSafeMalloc((void **)&dev_job.gather_arr, sizeof(int) * jobs_num);
  cudaSafeMalloc((void **)&dev_job.target_idx, sizeof(int) * jobs_num);
  cudaSafeMalloc((void **)&dev_job.image_dest, sizeof(float) * jobs_num);
  cudaSafeMalloc((void **)&dev_job.ray_pos, sizeof(float) * jobs_num);
  cudaSafeMalloc((void **)&dev_job.ray_dir, sizeof(float) * jobs_num);

  return dev_job;
}

void free_device_job(job_t *dev_job)
{
  cudaSafeFree(dev_job->gather_arr);
  dev_job->gather_arr = NULL;
  cudaSafeFree(dev_job->target_idx);
  dev_job->target_idx = NULL;
  cudaSafeFree(dev_job->image_dest);
  dev_job->image_dest = NULL;
  cudaSafeFree(dev_job->ray_pos);
  dev_job->ray_pos = NULL;
  cudaSafeFree(dev_job->ray_dir);
  dev_job->ray_dir = NULL;
}


void copy_job_to_dev(job_t *dev_dest, job_t *host_src)
{
  dev_dest->image_width = host_src->image_width;
  dev_dest->image_height = host_src->image_height;
  dev_dest->pass_count = host_src->pass_count;
  int hc = calc_jobs(host_src->image_width * host_src->image_height);
  hipMemcpy(dev_dest->gather_arr, host_src->gather_arr, hc * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_dest->target_idx, host_src->target_idx, hc * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_dest->image_dest, host_src->image_dest, hc * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_dest->ray_pos, host_src->ray_pos, hc * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_dest->ray_dir, host_src->ray_dir, hc * sizeof(int), hipMemcpyHostToDevice);
}

void copy_job_to_host(job_t *host_dest, job_t *dev_src)
{
  host_dest->image_width = dev_src->image_width;
  host_dest->image_height = dev_src->image_height;
  host_dest->pass_count = dev_src->pass_count;
  int hc = calc_jobs(dev_src->image_width * dev_src->image_height);
  hipMemcpy(host_dest->gather_arr, dev_src->gather_arr, hc * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(host_dest->target_idx, dev_src->target_idx, hc * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(host_dest->image_dest, dev_src->image_dest, hc * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(host_dest->ray_pos, dev_src->ray_pos, hc * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(host_dest->ray_dir, dev_src->ray_dir, hc * sizeof(int), hipMemcpyDeviceToHost);
}
