#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "common.cuh"
#include "bmp.cuh"
#include "scene.cuh"
#include "trace.cuh"
#include "job.cuh"

#include <cstdio>
#include <iostream>
#include <stack>

__global__ void ray_kernel(job_t job, int depth, scene_t *scene)
{
	int uniq_id = threadIdx.x + blockIdx.x * blockDim.x;

	// All threads do work but some of the results will be discarded as they are only the padding in the arrays.

	int kernel_x = uniq_id % blockDim.x;
	int kernel_y = uniq_id / blockDim.x;

	trace_ray(
		&job.image_dest[uniq_id * 3],	// assign the result color array
		&job.ray_pos[uniq_id * 3],		// assign job ray position
		&job.ray_dir[uniq_id * 3],		// assign job ray direction
		depth,												// this shall stop the recursion
		scene);												// const scene
}

__global__ void forward_kernel(job_t old_job, job_t new_job)
{
	int uniq_id = threadIdx.x + blockIdx.x * blockDim.x;

	if(old_job.gather_arr[uniq_id])
	{
		int dest_id = old_job.target_idx[uniq_id];
		set_vec3(&new_job.ray_pos[dest_id * 3], &old_job.ray_pos[uniq_id * 3]);
		set_vec3(&new_job.ray_dir[dest_id * 3], &old_job.ray_dir[uniq_id * 3]);
	}
}

__global__ void pps_kernel(int *dest, int *src, int powerof2Minus1)
{
	int uniq_id = threadIdx.x + blockIdx.x * blockDim.x;
	dest[uniq_id] = src[uniq_id - powerof2Minus1] + src[uniq_id];
}

static void do_pps(int *arr, int size)
{
	int d_max = ceil_log2(size);
	int *temp = NULL;
	cudaSafeMalloc(&temp, sizeof(int) * size);
	for(int d = 1; d <= d_max; d++)
	{
		pps_kernel<<< BLOCKS_PER_JOB(size), THREADS_PER_BLOCK >>>(temp, arr, pow2(d - 1));
		int *swap = temp;
		temp = arr;
		arr = swap;
	}
	if(d_max & 1)
	{
		hipMemcpy(arr, temp, size * sizeof(int), hipMemcpyDeviceToDevice);
	}

	cudaSafeFree(temp);
}

static int ray_step(job_t dev_job, scene_t *scene, int depth)
{
	int size = calc_jobs(dev_job.image_width * dev_job.image_height);
	assert(size > 0);
	ray_kernel<<< BLOCKS_PER_JOB(size), size % THREADS_PER_BLOCK >>>(dev_job, depth, scene);
	int next_size = 0;
	do_pps<<< BLOCKS_PER_JOB(size), size % THREADS_PER_BLOCK >>>(dev_job.target_idx, size);
	hipMemcpy(&dest_size, &dev_job.target_idx[size - 1], sizeof(int), hipMemcpyDeviceToHost);
	return next_size;
}

void main_loop(job_t host_job, scene_t *scene)
{
	int depth = 0;

	std::stack<job_t> jobs;

	job_t temp_job = host_job;
	job_t curr_job = allocate_device_job(temp_job);

	//buildup
	while(depth < 4)
	{
		int next_size = step(curr_job, scene, depth);
		if(next_size)
		{
			int size = calc_jobs(next_size);
			temp_job.image_width = next_size % THREADS_PER_BLOCK;
			temp_job.image_height = next_size / THREADS_PER_BLOCK;
			temp_job = allocate_device_job(temp_job);
			forward_kernel<<< BLOCKS_PER_JOB(size), size % THREADS_PER_BLOCK >>>(temp_job, curr_job);
			jobs.push(curr_job);
			curr_job = temp_job;
		}
		else
		{
			break;
		}
		depth++;
	}

	//unwind
	// TODO: unwind & merge, free
}
