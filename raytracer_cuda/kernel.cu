#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "bmp.cuh"
#include "scene.cuh"
#include "trace.cuh"

#include <cstdio>
#include <iostream>

#define TEST_WIDTH 1024
#define TEST_HEIGHT 1024

__global__ void ray_kernel(float *result_image, int divB, int sizeB, int ws, int hs, int width, int height, scene_t device_scene)
{
	int block_index = threadIdx.x + blockDim.x * threadIdx.y;

	int column = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;

	int x = row;
	int y = column;
	//trace_rect(result_image, x, y, ws, hs, width, height, &device_scene);
	float *color_offset = &result_image[(y * width + x) * 3];
	color_offset[0] = column / 1000.0f;
	color_offset[1] = row / 1000.0f;
	color_offset[2] = 0;
}

int main()
{   
	int size = TEST_WIDTH * TEST_HEIGHT * 3;
	float *host_result_image = (float *)malloc(sizeof(float) * size);
	float *cuda_result_image;
	if (hipMalloc(&cuda_result_image, sizeof(float)* size) != hipSuccess)
	{
		std::cerr << "Cannot allocate memory for result image on device!" << std::endl;
		exit(1);
	}

	init_scene("sample/sample", TEST_WIDTH, TEST_HEIGHT);

	std::cout << "[Prep] >> Done." << std::endl;

	int ws = TEST_WIDTH / TEST_WIDTH;
	int hs = TEST_HEIGHT / TEST_HEIGHT;

	ray_kernel << < TEST_WIDTH, TEST_HEIGHT >> >(cuda_result_image, TEST_WIDTH, TEST_HEIGHT, ws, hs, TEST_WIDTH, TEST_HEIGHT, dev_scene);
	hipDeviceSynchronize();

	//trace_all(TEST_WIDTH, TEST_HEIGHT, test);
	clean_scene();

	if (hipMemcpy(host_result_image, cuda_result_image, sizeof(float)* size, hipMemcpyDeviceToHost) != hipSuccess)
	{
		std::cerr << "Cannot copy result image from device!" << std::endl;
		exit(1);
	}

	FILE *file = fopen("test.bmp", "wb+");
	srand((unsigned int) time(NULL));
	if (file)
	{
		write_bmp(file, host_result_image, TEST_WIDTH, TEST_HEIGHT);
		fflush(file);
		fclose(file);
	}
	else
	{
		fprintf(stderr, "File could not be opened!\n");
	}
	free(host_result_image);
	hipFree(cuda_result_image);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	std::cout << "All done, press any key to exit..." << std::endl;
	std::cin.get();

    return 0;
}
