#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "bmp.cuh"
#include "scene.cuh"
#include "trace.cuh"

#include <cstdio>
#include <iostream>

#define TEST_WIDTH 1280
#define TEST_HEIGHT 1024

__global__ void ray_kernel(float *result_image, int divB, int sizeB, int ws, int hs, int width, int height)
{
	int x = 0;
	int y = 0;
	trace_rect(result_image, x, y, ws, hs, width, height);
}

int main()
{   
	int size = TEST_WIDTH * TEST_HEIGHT * 3;
	float *host_result_image = (float *)malloc(sizeof(float) * size);
	float *cuda_result_image;
	if (hipMalloc(&cuda_result_image, sizeof(float)* size) != hipSuccess)
	{
		std::cerr << "Cannot allocate memory for result image on device!" << std::endl;
		exit(1);
	}

	init_scene("sample", TEST_WIDTH, TEST_HEIGHT);

	std::cout << "[Prep] >> Done." << std::endl;

	int ws = TEST_WIDTH / 128;
	int hs = TEST_HEIGHT / 128;

	ray_kernel <<< 128, 128 >>>(cuda_result_image, 128, 128, ws, hs, TEST_WIDTH, TEST_HEIGHT);

	//trace_all(TEST_WIDTH, TEST_HEIGHT, test);
	clean_scene();

	if (hipMemcpy(host_result_image, cuda_result_image, sizeof(float)* size, hipMemcpyDeviceToHost) != hipSuccess)
	{
		std::cerr << "Cannot copy result image from device!" << std::endl;
		exit(1);
	}

	FILE *file = fopen("test.bmp", "wb+");
	srand((unsigned int) time(NULL));
	if (file)
	{
		write_bmp(file, host_result_image, TEST_WIDTH, TEST_HEIGHT);
		fflush(file);
		fclose(file);
	}
	else
	{
		fprintf(stderr, "File could not be opened!\n");
	}
	free(host_result_image);
	hipFree(cuda_result_image);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	std::cout << "All done, press any key to exit..." << std::endl;
	std::cin.get();

    return 0;
}
